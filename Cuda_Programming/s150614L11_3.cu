#include "hip/hip_runtime.h"
#include "kernel.h"
#include "Math.h"
#define TX 32
#define TY 32


#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

// CUDA event ��ü�� ����Ͽ� Ŀ�� ����ð� ����
void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }


__global__ void distanceKernel(uchar4 *d_out, int w, int h, int2 pos)
{
	const int c = blockIdx.x * blockDim.x + threadIdx.x;	//����� id * ����� �� + ��� �������� thread id
	const int r = blockIdx.y * blockDim.y + threadIdx.y;	//����� id * ����� �� + ��� �������� thread id

	//���ϴ� ���� ���� �����ʹ� ������� �ʴ´�.
	if ((c >= w) || (r >= h)) return;
	const int i = r * w + c;	//��ü thread ������ ����(id)
	
	int tmp = 255 - int(sqrt(double((pos.x - c)*(pos.x - c) + (pos.y - r)*(pos.y - r))));
	if (tmp < 0) tmp = 0;
	d_out[i].x = tmp; 		//R
	d_out[i].y = tmp; 		//G
	d_out[i].z = 0; 		//B
	d_out[i].w = 255; 	//A (������)	
}

//Ŀ���� ȣ���ϴ� CPU �Լ�. 
float kernelLauncher(uchar4 *d_out, int w, int h, int2 pos) {
	float time=0;
	//����� ũ��. ���ΰ� TX��, ���ΰ� TY��
	const dim3 blockSize(TX, TY);

	//grid, �� thread block�� ��. ����� grid.x * grid.y �� ����.
	const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);

	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();
	distanceKernel << <gridSize, blockSize >> > (d_out, w, h, pos);
	CHECK_TIME_END_GPU(time);
	CHECK_TIME_DEST_GPU();

	return time;
}